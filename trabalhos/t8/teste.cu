#include "hip/hip_runtime.h"
#include <cstdlib>
#include <sys/time.h>
#include <math.h>
#include "wave.h"

__global__
void calculateFrame(unsigned char* pic, int width) {
    int index = threadIdx.x;
    for (int row = 0; row < width; row++) {
        for (int col = 0; col < width; col++) {
            float fx = col - 1024/2;
            float fy = row - 1024/2;
            float d = sqrtf( fx * fx + fy * fy );
            unsigned char color = (unsigned char) (160.0f + 127.0f * cos(d/10.0f - index/7.0f) / (d/50.0f + 1.0f));
            pic[index * width * width + row * width + col] = (unsigned char) color;
        }
    }
}

int main(int argc, char *argv[]){
    // check command line
    if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
    int width = atoi(argv[1]);
    if (width < 100) {fprintf(stderr, "error: frame_width must be at least 100\n"); exit(-1);}
    int frames = atoi(argv[2]);
    if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
    printf("computing %d frames of %d by %d picture\n", frames, width, width);

    // allocate picture array
    unsigned char* pic;
    hipMallocManaged(&pic, frames*width*width*sizeof(unsigned char));

    // start time
    timeval start, end;
    gettimeofday(&start, NULL);

    calculateFrame<<<1, frames>>>(pic, width);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // end time
    gettimeofday(&end, NULL);
    double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
    printf("compute time: %.4f s\n", runtime);

    // verify result by writing frames to BMP files
    if ((width <= 256) && (frames <= 100)) {
        for (int frame = 0; frame < frames; frame++) {
            char name[32];
            sprintf(name, "wave%d.bmp", frame + 1000);
            writeBMP(width, width, &pic[frame * width * width], name);
        }
    }

    hipFree(pic);
    return 0;
}